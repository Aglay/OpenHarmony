#include "hip/hip_runtime.h"
/*
*
* Copyright (C) 2019. Huawei Technologies Co., Ltd. All rights reserved.
*
*/

#include <vector>

#include "caffe/layers/conv_layer.hpp"
#include "caffe/util/im2col.hpp"
#include "caffe/opt/layers/opt_conv_layer.hpp"
#include "caffe/opt/utils/math_funcs.hpp"

namespace caffe {

#ifdef USE_CUDNN
__global__ void sync_opt_conv_groups() {}
#endif

template <typename Dtype>
void OptConvolutionLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
  int index = 0;
  if (this->bias_term_) {
    index += 1;
  }
  
  vector<shared_ptr<Blob<Dtype> > > weights;
  if (this->blobs_.size() != index+7)
  {
    printf("invalid blobs_ size!\n");
    return;
  }
  weights.push_back(this->blobs_[0]);
  weights.push_back(this->blobs_[index+1]);
  weights.push_back(this->blobs_[index+2]);
  weights.push_back(this->blobs_[index+3]);
  weights.push_back(this->blobs_[index+4]);
  weights.push_back(this->blobs_[index+5]);
  weights.push_back(this->blobs_[index+6]);

  weights.push_back(this->inf_bottom_);
  vector<Blob<Dtype>* > datas;
  datas.push_back(top[0]);
  datas.push_back(bottom[0]);
  int is_train = 0;
  if(this->phase_ == TRAIN)
  {
    is_train = 1;
    this->opt_input_or_weight_ = OPT_WEIGHT;
    this->Opt_gpu(weights, datas, is_train, OPT_CONV);
  }

  
  this->opt_input_or_weight_ = OPT_INPUT;
  this->Opt_gpu(weights, datas, is_train, OPT_CONV);  
  
  
  const Dtype* weight = this->blobs_[index+1]->gpu_data();  //quantized weight
#ifdef USE_CUDNN
  for (int i = 0; i < bottom.size(); ++i) {
    const Dtype* bottom_data = this->inf_bottom_->gpu_data();
    Dtype* top_data = top[i]->mutable_gpu_data();   
    // Forward through cuDNN in parallel over groups.
    for (int g = 0; g < this->group_; g++) {
      // Filters.
      CUDNN_CHECK(hipdnnConvolutionForward(handle_[g],
            cudnn::dataType<Dtype>::one,
            bottom_descs_[i], bottom_data + bottom_offset_ * g,
            filter_desc_, weight + this->weight_offset_ * g,
            conv_descs_[i],
            fwd_algo_[i], workspace[g], workspace_fwd_sizes_[i],
            cudnn::dataType<Dtype>::zero,
            top_descs_[i], top_data + top_offset_ * g));

      // Bias.
      if (this->bias_term_) {
        const Dtype* bias_data = this->blobs_[1]->gpu_data();
        CUDNN_CHECK(hipdnnAddTensor(handle_[g],
              cudnn::dataType<Dtype>::one,
              bias_desc_, bias_data + bias_offset_ * g,
              cudnn::dataType<Dtype>::one,
              top_descs_[i], top_data + top_offset_ * g));
      }
    }

    // Synchronize the work across groups, each of which went into its own
    // stream, by launching an empty kernel into the default (null) stream.
    // NOLINT_NEXT_LINE(whitespace/operators)
    sync_opt_conv_groups<<<1, 1>>>();
  }
#else
  for (int i = 0; i < bottom.size(); ++i) {
    const Dtype* bottom_data = this->inf_bottom_->gpu_data();
    Dtype* top_data = top[i]->mutable_gpu_data();
    for (int n = 0; n < this->num_; ++n) {
      this->forward_gpu_gemm(bottom_data + n * this->bottom_dim_, weight,
          top_data + n * this->top_dim_);
      if (this->bias_term_) {
        const Dtype* bias = this->blobs_[1]->gpu_data();
        this->forward_gpu_bias(top_data + n * this->top_dim_, bias);
      }
    }
  }
#endif
}

template <typename Dtype>
void OptConvolutionLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {

  int index = 0;
  if (this->bias_term_) 
    index += 1;

#ifdef USE_CUDNN
  const Dtype* weight = NULL;
  Dtype* weight_diff = NULL;
  if (this->param_propagate_down_[0]) {
    // weight = this->blobs_[0]->gpu_data(); //full weight
    weight = this->blobs_[index+1]->gpu_data(); //quantized weight
    weight_diff = this->blobs_[0]->mutable_gpu_diff();
  }
  Dtype* bias_diff = NULL;
  if (this->bias_term_ && this->param_propagate_down_[1]) {
    bias_diff = this->blobs_[1]->mutable_gpu_diff();
  }
  for (int i = 0; i < top.size(); ++i) {
    const Dtype* top_diff = top[i]->gpu_diff();
    // Backward through cuDNN in parallel over groups and gradients.
    for (int g = 0; g < this->group_; g++) {
      // Gradient w.r.t. bias.
      if (this->bias_term_ && this->param_propagate_down_[1]) {
        CUDNN_CHECK(hipdnnConvolutionBackwardBias(handle_[0*this->group_ + g],
              cudnn::dataType<Dtype>::one,
              top_descs_[i],  top_diff + top_offset_ * g,
              cudnn::dataType<Dtype>::one,
              bias_desc_, bias_diff + bias_offset_ * g));
      }

      // Gradient w.r.t. weights.
      if (this->param_propagate_down_[0]) {
        const Dtype* bottom_data = bottom[i]->gpu_data();
        CUDNN_CHECK(hipdnnConvolutionBackwardFilter(
              handle_[1*this->group_ + g],
              cudnn::dataType<Dtype>::one,
              bottom_descs_[i], bottom_data + bottom_offset_ * g,
              top_descs_[i],    top_diff + top_offset_ * g,
              conv_descs_[i],
              bwd_filter_algo_[i], workspace[1*this->group_ + g],
              workspace_bwd_filter_sizes_[i],
              cudnn::dataType<Dtype>::one,
              filter_desc_, weight_diff + this->weight_offset_ * g));
      }

      // Gradient w.r.t. bottom data.
      if (propagate_down[i]) {
        if (weight == NULL) {
          // weight = this->blobs_[0]->gpu_data();  //full weight
          weight = this->blobs_[index+1]->gpu_data(); //quantized weight
        }
        Dtype* bottom_diff = bottom[i]->mutable_gpu_diff();
        CUDNN_CHECK(hipdnnConvolutionBackwardData(
              handle_[2*this->group_ + g],
              cudnn::dataType<Dtype>::one,
              filter_desc_, weight + this->weight_offset_ * g,
              top_descs_[i], top_diff + top_offset_ * g,
              conv_descs_[i],
              bwd_data_algo_[i], workspace[2*this->group_ + g],
              workspace_bwd_data_sizes_[i],
              cudnn::dataType<Dtype>::zero,
              bottom_descs_[i], bottom_diff + bottom_offset_ * g));
      }
    }

    // Synchronize the work across groups, each of which went into its own
    // stream, by launching an empty kernel into the default (null) stream.
    // NOLINT_NEXT_LINE(whitespace/operators)
    sync_opt_conv_groups<<<1, 1>>>();
  }
#else
  const Dtype* weight = this->blobs_[index+1]->gpu_data(); //quantized weight

  Dtype* weight_diff = this->blobs_[0]->mutable_gpu_diff();

  for (int i = 0; i < top.size(); ++i) {
    const Dtype* top_diff = top[i]->gpu_diff();
    // Bias gradient, if necessary.
    if (this->bias_term_ && this->param_propagate_down_[1]) {
      Dtype* bias_diff = this->blobs_[1]->mutable_gpu_diff();
      for (int n = 0; n < this->num_; ++n) {
        this->backward_gpu_bias(bias_diff, top_diff + n * this->top_dim_);
      }
    }
    if (this->param_propagate_down_[0] || propagate_down[i]) {
      const Dtype* bottom_data = bottom[i]->gpu_data();
      Dtype* bottom_diff = bottom[i]->mutable_gpu_diff();
      for (int n = 0; n < this->num_; ++n) {
        // gradient w.r.t. weight. Note that we will accumulate diffs.
        if (this->param_propagate_down_[0]) {
          this->weight_gpu_gemm(bottom_data + n * this->bottom_dim_,
              top_diff + n * this->top_dim_, weight_diff);
        }
        // gradient w.r.t. bottom data, if necessary.
        if (propagate_down[i]) {
          this->backward_gpu_gemm(top_diff + n * this->top_dim_, weight,
              bottom_diff + n * this->bottom_dim_);
        }
      }
    }
  }
#endif
  vector<shared_ptr<Blob<Dtype> > > weights;
  if (this->blobs_.size() != index+7)
  {
    printf("invalid blobs_ size!\n");
    return;
  }
  weights.push_back(this->blobs_[0]);
  weights.push_back(this->blobs_[index+4]);
  weights.push_back(this->blobs_[index+5]);
  vector<Blob<Dtype>* > datas;
  datas.push_back(top[0]);
  datas.push_back(bottom[0]);

  this->opt_input_or_weight_ = OPT_INPUT;
  this->Opt_BP_gpu(weights, datas);
}

INSTANTIATE_LAYER_GPU_FUNCS(OptConvolutionLayer);

}  // namespace caffe
